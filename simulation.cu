#include "hip/hip_runtime.h"
#include "cuda_util.cuh"
#include "cuda_reduction.cuh"
#include "cuda_for.cuh"
#include "cuda_random.cuh"

#include "simulation.h"

typedef Sim_Real Real;

#include <assert.h>
#ifndef ASSERT
    #define ASSERT(x)
#endif

thread_local static hipEvent_t _cuda_timer_start = NULL;
thread_local static hipEvent_t _cuda_timer_stop = NULL;

void cuda_timer_start()
{
    if(_cuda_timer_start == NULL || _cuda_timer_stop == NULL)
    {
        CUDA_TEST(hipEventCreate(&_cuda_timer_start));
        CUDA_TEST(hipEventCreate(&_cuda_timer_stop));
    }
    CUDA_TEST(hipEventRecord(_cuda_timer_start, 0));
}

double cuda_timer_stop()
{
    CUDA_TEST(hipEventRecord(_cuda_timer_stop, 0));
    CUDA_TEST(hipEventSynchronize(_cuda_timer_stop));

    float time = 0;
    CUDA_TEST(hipEventElapsedTime(&time, _cuda_timer_start, _cuda_timer_stop));
    return (double) time / 1000;
}

template <typename T>
void sim_modify_T(Real* device_memory, T* host_memory, size_t count, Sim_Modify modify)
{
    static T* static_device = NULL;
    static size_t static_size = 0;

    if(sizeof(Real) != sizeof(T))
    {
        if(static_size < count)
        {
            cuda_realloc_in_place((void**) &static_device, count*sizeof(T), static_size*sizeof(T), 0);
            static_size = count;
        }

        T* temp_device = static_device;
        if(modify == MODIFY_UPLOAD)
        {
            //Upload: host -> static -> device
            CUDA_DEBUG_TEST(hipMemcpy(temp_device, host_memory, count*sizeof(T), hipMemcpyHostToDevice));
            cuda_for(0, (int) count, [=]SHARED(int i){
                device_memory[i] = (Real) temp_device[i];
            });
        }
        else
        {
            //download: device -> static -> host
            cuda_for(0, (int) count, [=]SHARED(int i){
                temp_device[i] = (T) device_memory[i];
            });
            CUDA_DEBUG_TEST(hipMemcpy(host_memory, temp_device, count*sizeof(T), hipMemcpyDeviceToHost));
        }
    }
    else
    {
        if(modify == MODIFY_UPLOAD)
            CUDA_DEBUG_TEST(hipMemcpy(device_memory, host_memory, count*sizeof(T), hipMemcpyHostToDevice));
        else
            CUDA_DEBUG_TEST(hipMemcpy(host_memory, device_memory, count*sizeof(T), hipMemcpyDeviceToHost));
    }
}

extern "C" void sim_modify(void* device_memory, void* host_memory, size_t size, Sim_Modify modify)
{
    if(modify == MODIFY_UPLOAD)
        CUDA_DEBUG_TEST(hipMemcpy(device_memory, host_memory, size, hipMemcpyHostToDevice));
    else
        CUDA_DEBUG_TEST(hipMemcpy(host_memory, device_memory, size, hipMemcpyDeviceToHost));
}

extern "C" void sim_modify_float(Real* device_memory, float* host_memory, size_t count, Sim_Modify modify)
{   
    sim_modify_T(device_memory, host_memory, count, modify);
}

extern "C" void sim_modify_double(Real* device_memory, double* host_memory, size_t count, Sim_Modify modify)
{   
    sim_modify_T(device_memory, host_memory, count, modify);
}


extern "C" bool sim_mut_state_init(Sim_Mut_State* state, int32_t nx, int32_t ny)
{
    sim_mut_state_deinit(state);
    state->nx = nx;
    state->ny = ny;
    size_t bytes = (size_t) nx * (size_t) ny * sizeof(Real); 
    CUDA_TEST(hipMalloc(&state->rho, bytes));
    CUDA_TEST(hipMalloc(&state->ux, bytes));
    CUDA_TEST(hipMalloc(&state->uy, bytes));
    return true;
}
extern "C" bool sim_const_state_init(Sim_Const_State* state, int32_t nx, int32_t ny)
{
    sim_const_state_deinit(state);
    state->nx = nx;
    state->ny = ny;
    size_t bytes = (size_t) nx * (size_t) ny * sizeof(Real); 

    CUDA_TEST(hipMalloc(&state->flags, (size_t) nx * (size_t) ny * sizeof(Sim_Flags)));
    CUDA_TEST(hipMalloc(&state->set_rho, bytes));
    CUDA_TEST(hipMalloc(&state->set_ux, bytes));
    CUDA_TEST(hipMalloc(&state->set_uy, bytes));

    CUDA_TEST(hipMalloc(&state->set_dx_rho, bytes));
    CUDA_TEST(hipMalloc(&state->set_dx_ux, bytes));
    CUDA_TEST(hipMalloc(&state->set_dx_uy, bytes));

    CUDA_TEST(hipMalloc(&state->set_dy_rho, bytes));
    CUDA_TEST(hipMalloc(&state->set_dy_ux, bytes));
    CUDA_TEST(hipMalloc(&state->set_dy_uy, bytes));
    return true;
}

extern "C" void sim_mut_state_deinit(Sim_Mut_State* state)
{
    CUDA_TEST(hipFree(state->rho));
    CUDA_TEST(hipFree(state->ux));
    CUDA_TEST(hipFree(state->uy));
    memset(state, 0, sizeof *state);
}
extern "C" void sim_const_state_deinit(Sim_Const_State* state)
{
    CUDA_TEST(hipFree(state->flags));

    CUDA_TEST(hipFree(state->set_rho));
    CUDA_TEST(hipFree(state->set_ux));
    CUDA_TEST(hipFree(state->set_uy));

    CUDA_TEST(hipFree(state->set_dx_rho));
    CUDA_TEST(hipFree(state->set_dx_ux));
    CUDA_TEST(hipFree(state->set_dx_uy));

    CUDA_TEST(hipFree(state->set_dy_rho));
    CUDA_TEST(hipFree(state->set_dy_ux));
    CUDA_TEST(hipFree(state->set_dy_uy));
    memset(state, 0, sizeof *state);
}

struct Vars {
    Real rho;
    Real ux;
    Real uy;
};

extern "C" double sim_step(Sim_Mut_State* next, const Sim_Mut_State* prev, Sim_Const_State* const_state, Sim_Params params)
{
    csize nx = next->nx;
    csize ny = next->ny;

    Real* prev_rhos = prev->rho;
    Real* prev_uxs = prev->ux;
    Real* prev_uys = prev->uy;

    Real* next_rhos = next->rho;
    Real* next_uxs = next->ux;
    Real* next_uys = next->uy;

    Sim_Real* set_rho = const_state->set_rho;
    Sim_Real* set_ux = const_state->set_ux;
    Sim_Real* set_uy = const_state->set_uy;

    Sim_Real* set_dx_rho = const_state->set_dx_rho;
    Sim_Real* set_dx_ux = const_state->set_dx_ux;
    Sim_Real* set_dx_uy = const_state->set_dx_uy;

    Sim_Real* set_dy_rho = const_state->set_dy_rho;
    Sim_Real* set_dy_ux = const_state->set_dy_ux;
    Sim_Real* set_dy_uy = const_state->set_dy_uy;

    Sim_Flags* flags = const_state->flags;
    Real dt = params.dt;
    Real dx = params.region_width / nx;
    Real dy = params.region_height / ny;
    Real lambda = params.dynamic_viscosity;
    Real mu = params.second_viscosity;
    
    Real R_spec = 287.052874;
    Real T = params.temperature;
    //TODO: try the second version of tiled for as well - should be faster more kernels where most of the time
    // is spent computing.

    #define PRINT_F(x) printf(#x " = %e\n", (double) (x))

    #define PRINT_VARS(v) printf(#v " = {rho:%e ux:%e uy:%e}\n", (v).rho, (v).ux, (v).uy)

    //Precaclulate first order derivations
    cuda_tiled_for_2D<1, 1, Vars>(0, 0, nx, ny, 
        [=]SHARED(csize x, csize y, csize nx, csize ny, csize rx, csize ry){
            if(0 <= x && x < nx && 0 <= y && y < ny) {
                csize i = x + y*nx;
                Sim_Flags cflags = flags[i];
                
                return Vars{
                    cflags & SIM_SET_RHO ? set_rho[i] : prev_rhos[i],
                    cflags & SIM_SET_UX ? set_ux[i] : prev_uxs[i],
                    cflags & SIM_SET_UY ? set_uy[i] : prev_uys[i],
                };
            }
            else
                return Vars{0};
        },
        [=]SHARED(csize x, csize y, csize tx, csize ty, csize tile_size_x, csize tile_size_y, Vars* __restrict__ shared){
            csize i = x + y*nx;
            Sim_Flags cflags = flags[i];
            Vars vc = shared[tx + ty*tile_size_x];
            
            // next_rhos[i] = vc.rho;
            // next_uxs[i] = vc.ux;
            // next_uys[i] = vc.uy;

            // return;
            Vars vn = shared[tx + (ty+1)*tile_size_x];
            Vars vs = shared[tx + (ty-1)*tile_size_x];
            Vars ve = shared[(tx+1) + ty*tile_size_x];
            Vars vw = shared[(tx-1) + ty*tile_size_x];

            Vars vne = shared[(tx+1) + (ty+1)*tile_size_x];
            Vars vnw = shared[(tx-1) + (ty+1)*tile_size_x];
            Vars vse = shared[(tx+1) + (ty-1)*tile_size_x];
            Vars vsw = shared[(tx-1) + (ty-1)*tile_size_x];

            //TODO
            Real Fx = 0; 
            Real Fy = 0;

            Real rho = vc.rho;
            Real ux = vc.ux;
            Real uy = vc.uy;

            Real dx_rho = (ve.rho - vw.rho)/(2*dx);
            Real dx_ux = (ve.ux - vw.ux)/(2*dx);
            Real dx_uy = (ve.uy - vw.uy)/(2*dx);

            Real dy_rho = (vn.rho - vs.rho)/(2*dy);
            Real dy_ux = (vn.ux - vs.ux)/(2*dy);
            Real dy_uy = (vn.uy - vs.uy)/(2*dy);

            Real dxx_ux = (ve.ux + 2*vc.ux - vw.ux)/(dx*dx);
            Real dxx_uy = (ve.uy + 2*vc.uy - vw.uy)/(dx*dx);

            Real dyy_ux = (vn.ux + 2*vc.ux - vs.ux)/(dy*dy);
            Real dyy_uy = (vn.uy + 2*vc.uy - vs.uy)/(dy*dy);

            Real dxy_ux = (vne.ux - vnw.ux - vse.ux + vsw.ux)/(4*dx*dy);
            Real dxy_uy = (vne.uy - vnw.uy - vse.uy + vsw.uy)/(4*dx*dy);

            #if 0
            Real ux_dx_ux = ux >= 0 ? ux*(vc.ux - vw.ux)/dx : ux*(ve.ux - vc.ux)/dx;
            Real ux_dx_uy = ux >= 0 ? ux*(vc.uy - vw.uy)/dx : ux*(ve.uy - vc.uy)/dx;
            Real uy_dy_ux = uy >= 0 ? uy*(vc.ux - vs.ux)/dy : uy*(vn.ux - vc.ux)/dy;
            Real uy_dy_uy = uy >= 0 ? uy*(vc.uy - vs.uy)/dy : uy*(vn.uy - vc.uy)/dy;

            Real ux_dx_rho = ux >= 0 ? ux*(vc.rho - vw.rho)/dx : ux*(ve.rho - vc.rho)/dx;
            Real uy_dy_rho = uy >= 0 ? uy*(vc.rho - vs.rho)/dy : uy*(vn.rho - vc.rho)/dy;
            #else

            Real ux_dx_ux = true ? ux*(vc.ux - vw.ux)/dx : ux*(ve.ux - vc.ux)/dx;
            Real ux_dx_uy = true ? ux*(vc.uy - vw.uy)/dx : ux*(ve.uy - vc.uy)/dx;
            Real uy_dy_ux = true ? uy*(vc.ux - vs.ux)/dy : uy*(vn.ux - vc.ux)/dy;
            Real uy_dy_uy = true ? uy*(vc.uy - vs.uy)/dy : uy*(vn.uy - vc.uy)/dy;

            Real ux_dx_rho = true ? ux*(vc.rho - vw.rho)/dx : ux*(ve.rho - vc.rho)/dx;
            Real uy_dy_rho = true ? uy*(vc.rho - vs.rho)/dy : uy*(vn.rho - vc.rho)/dy;
            #endif

            if(cflags) {
                if(cflags & SIM_SET_DX_UX) dx_ux = set_dx_ux[i];
                if(cflags & SIM_SET_DX_UY) dx_uy = set_dx_uy[i];
                if(cflags & SIM_SET_DY_UX) dy_ux = set_dy_ux[i];
                if(cflags & SIM_SET_DY_UY) dy_uy = set_dy_uy[i];

                if(cflags & SIM_SET_DX_UX) ux_dx_ux = ux*dx_ux;
                if(cflags & SIM_SET_DX_UY) ux_dx_uy = ux*dx_uy;
                if(cflags & SIM_SET_DY_UX) uy_dy_ux = uy*dy_ux;
                if(cflags & SIM_SET_DY_UY) uy_dy_uy = uy*dy_uy;

                if(cflags & SIM_SET_DX_RHO) dx_rho = set_dx_rho[i];
                if(cflags & SIM_SET_DY_RHO) dy_rho = set_dy_rho[i];
                if(cflags & SIM_SET_DX_RHO) ux_dx_rho = ux*dx_rho;
                if(cflags & SIM_SET_DY_RHO) uy_dy_rho = uy*dy_rho;
            }

            #if 1
            Real dt_rho = -(ux_dx_rho + uy_dy_rho) + rho*(dx_ux + dy_uy);
            Real dt_ux = -(ux_dx_ux + uy_dy_ux) - R_spec*T/rho*dx_rho + Fx
                + 1/rho*((lambda + 2*mu)*dxx_ux + lambda*dxy_uy + mu*dyy_ux + mu*dxy_uy);
            Real dt_uy = -(ux_dx_uy + uy_dy_uy) - R_spec*T/rho*dy_rho + Fy
                + 1/rho*((lambda + 2*mu)*dyy_uy + lambda*dxy_ux + mu*dxx_uy + mu*dxy_ux);
            #else
            Real dt_rho = -(ux_dx_rho + uy_dy_rho) + rho*(dx_ux + dy_uy);
            Real dt_ux = -(ux_dx_ux + uy_dy_ux) - R_spec*T/rho*dx_rho + Fx;
            Real dt_uy = -(ux_dx_uy + uy_dy_uy) - R_spec*T/rho*dy_rho + Fy;
            #endif

            if(0)
            if(x == 1 && y == 2) {
                PRINT_VARS(vc);
                PRINT_VARS(vn);
                PRINT_VARS(vs);
                PRINT_VARS(ve);
                PRINT_VARS(vw);
                PRINT_VARS(vne);
                PRINT_VARS(vnw);
                PRINT_VARS(vse);
                PRINT_VARS(vsw);

                PRINT_F(dx);
                PRINT_F(dy);
                PRINT_F(ux);
                PRINT_F(uy);
                PRINT_F(dx_ux);
                PRINT_F(dx_uy);
                PRINT_F(dy_ux);
                PRINT_F(dy_uy);
                PRINT_F(dx_rho);
                PRINT_F(dy_rho);
                PRINT_F(dxx_ux);
                PRINT_F(dxx_uy);
                PRINT_F(dyy_ux);
                PRINT_F(dyy_uy);
                PRINT_F(dxy_ux);
                PRINT_F(dxy_uy);
                PRINT_F(ux_dx_ux);
                PRINT_F(ux_dx_uy);
                PRINT_F(uy_dy_ux);
                PRINT_F(uy_dy_uy);
                PRINT_F(ux_dx_rho);
                PRINT_F(uy_dy_rho);

                PRINT_F(dt_rho);
                PRINT_F(dt_ux);
                PRINT_F(dt_uy);
            }

            if(cflags) {
                if(cflags & SIM_SET_RHO) dt_rho = 0;
                if(cflags & SIM_SET_UX) dt_ux = 0;
                if(cflags & SIM_SET_UY) dt_uy = 0;
            }

            // dt_rho = 0;
            next_rhos[i] = rho + dt*dt_rho;
            next_uxs[i] = ux + dt*dt_ux;
            next_uys[i] = uy + dt*dt_uy;
        }
    );

    return dt;
}

extern "C" bool sim_make_flow_vertices(Sim_Flow_Vertex* vertices, Real* uxs, Real* uys, Draw_Lines_Config config)
{
    csize nx = (csize) config.nx;
    csize ny = (csize) config.ny;
    csize pix_size = config.pix_size;
    float pix_sizef = config.pix_size;

    cuda_for_2D(0, 0, nx/pix_size, ny/pix_size, [=]SHARED(csize xi, csize yi){
        xi *= pix_size;
        yi *= pix_size;
        csize i = xi + yi*nx;

        Real realux = 0;
        Real realuy = 0;
        for(csize ox = 0; ox < pix_size; ox++)
            for(csize oy = 0; oy < pix_size; oy++)
            {
                csize ic = (xi + ox) + (yi + oy)*nx;
                realux += uxs[ic];
                realuy += uys[ic];
            }
        
        float ux = (float) realux / (pix_sizef*pix_sizef);
        float uy = (float) realuy / (pix_sizef*pix_sizef);

        //normalize direction
        float len = hypotf(ux, uy);
        if(len > 0) {
            ux /= len;
            uy /= len;
        }

        //calculate variability
        #if 0
        Real variability = 0;
        for(csize ox = 0; ox < pix_size; ox++)
            for(csize oy = 0; oy < pix_size; oy++)
            {
                csize ic = (xi + ox) + (yi + oy)*nx;
                variability += fabs(ux*uxs[ic] + ux*uys[ic]);
            }
        variability = 1 - variability/(pix_sizef*pix_sizef);
        #endif

        float x = (xi + pix_sizef/2)*config.dx*2 - 1;
        float y = (yi + pix_sizef/2)*config.dy*2 - 1;

        float scaled_len = len*config.scale;
        if(scaled_len < config.min_size)
            scaled_len = config.min_size;
        if(scaled_len > config.max_size)
            scaled_len = config.max_size;
        if(len == 0)
            len = 1;

        float px = uy;
        float py = -ux;

        float ex = ux*scaled_len + x;
        float ey = uy*scaled_len + y;

        float v1x = x + px*config.width_i0;
        float v1y = y + py*config.width_i0;

        float v2x = x - px*config.width_i0;
        float v2y = y - py*config.width_i0;

        float v3x = ex + px*config.width_i1;
        float v3y = ey + py*config.width_i1;

        float v4x = ex - px*config.width_i1;
        float v4y = ey - py*config.width_i1;

        vertices[i*6+0] = Sim_Flow_Vertex{v1x, v1y, config.rgba_i0};
        vertices[i*6+1] = Sim_Flow_Vertex{v2x, v2y, config.rgba_i0};
        vertices[i*6+2] = Sim_Flow_Vertex{v3x, v3y, config.rgba_i1};
        vertices[i*6+3] = Sim_Flow_Vertex{v2x, v2y, config.rgba_i0};
        vertices[i*6+4] = Sim_Flow_Vertex{v3x, v3y, config.rgba_i1};
        vertices[i*6+5] = Sim_Flow_Vertex{v4x, v4y, config.rgba_i1};
    });
    return true;
}


extern "C" bool sim_run_tests()
{
    #ifdef TEST_CUDA_FOR_IMPL
    test_tiled_for((uint64_t) clock_ns());
    test_tiled_for_2D((uint64_t) clock_ns());
    #endif
    #ifdef TEST_CUDA_REDUCTION_IMPL
    test_reduce((uint64_t) clock_ns());
    #endif

    return true;
}